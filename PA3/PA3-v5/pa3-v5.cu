// Template for Programming Assignment 3
// Use "module load cuda" to enable compilation with the Nvidia C compiler nvcc
// Use "nvcc -O3" to compile code; this can be done even on an OSC login node (does not have a GPU)
// To execute compiled code, you must either use a batch submission to run on a node with GPU
// or obtain an interactive GPU-node by using: qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1 -A PAS1488


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-8
#define n (1024)
// Change n to 1024 for final testing; 
// #define n (256)
// n is set to 256 since execution time of single thread template version is excessive
#define SIZE (32)
void init(void);
void ref(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *C);
double rtclock(void);

double a[n][n],b[n][n],c[n][n],cref[n][n];

int main(){

  double clkbegin, clkend, t;
  double *Ad,*Bd,*Cd;
  int size;

  printf("Matrix Size = %d\n",n);

  init();
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; cref[n/2][n/2-1] = %f; \n",
    2.0*n*n*n/t/1e9,t,cref[n/2][n/2-1]);

  
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Cd,size);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);

  dim3 threads(SIZE, SIZE);
  dim3 grid(n / threads.x / 2, n / threads.y / 2);

  clkbegin = rtclock();
  test_kernel<<<grid, threads>>>(n,Ad,Bd,Cd);
  if (hipDeviceSynchronize() != hipSuccess) 
    printf ("Error return for test_kernel: Was execution done on a node with a GPU?\n");
  else
  {
   clkend = rtclock();
   t = clkend-clkbegin;
   hipMemcpy(c,Cd,size,hipMemcpyDeviceToHost);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);
   printf("GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; c[n/2][n/2-1] = %f; \n",
     2.0*n*n*n/t/1e9,t,c[n/2][n/2-1]);
   printf("Correctness Check for GPU solution:\n");
   compare(n, (double *) c,(double *) cref);
 }
}

__global__ void test_kernel(int N, double *A, double *B, double *C)
{
  int i,j,k;
// Template version uses only one thread, which does all the work
// This must be changed (and the launch parameters) to exploit GPU parallelism
// You can make any changes; only requirement is that correctness test passes

  int tx = threadIdx.x, ty = threadIdx.y;
  i = blockIdx.x * blockDim.x * 2 + tx;
  j = blockIdx.y * blockDim.y * 2 + ty;

  __shared__ double 
    Bb0[SIZE * SIZE], Bb1[SIZE * SIZE],
    Ab0[SIZE * SIZE], Ab1[SIZE * SIZE];


  double sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
  for (int ks = 0; ks < N; ks += SIZE) {
    Bb0[ty * SIZE + tx] = B[(ks + tx) * N + j];
    Bb1[ty * SIZE + tx] = B[(ks + tx) * N + j + SIZE];
    Ab0[ty * SIZE + tx] = A[(ks + ty) * N + i];
    Ab1[ty * SIZE + tx] = A[(ks + ty) * N + i + SIZE];
    __syncthreads();
    for (k = 0; k < SIZE; k++) {
      double Atmp0 = Ab0[k * SIZE + tx], Atmp1 = Ab1[k * SIZE + tx];
      double Btmp0 = Bb0[ty * SIZE + k], Btmp1 = Bb1[ty * SIZE + k];
      sum0 += Atmp0 * Btmp0;  // C[j][i]
      sum1 += Atmp1 * Btmp0;  // C[j][i + SIZE]
      sum2 += Atmp0 * Btmp1;  // C[j + SIZE][i]
      sum3 += Atmp1 * Btmp1;  // C[j + SIZE][i + SIZE]
    }
    __syncthreads();
  }
  C[j * N + i] = sum0;
  C[j * N + SIZE + i] = sum1;
  C[(j + SIZE) * N + i] = sum2;
  C[(j + SIZE) * N + SIZE + i] = sum3;
}

void ref(void)
{
  int i,j,k;

  for (j=0;j<n;j++)
   for (k=0;k<n;k++)
    for (i=0;i<n;i++)
      cref[j][i] += a[k][i]*b[k][j];
  }

  void init(void)
  {
    int i,j;
    for(i=0;i<n;i++)
     for(j=0;j<n;j++) 
       { c[i][j] = 0.0; 
         cref[i][j] = 0.0; 
         a[i][j] = drand48();
         b[i][j] = drand48();
       }
     }

     void compare(int N, double *wref, double *w)
     {
      double maxdiff,this_diff;
      int numdiffs;
      int i,j;
      numdiffs = 0;
      maxdiff = 0;
      for (i=0;i<N;i++)
       for (j=0;j<N;j++)
       {
         this_diff = wref[i*N+j]-w[i*N+j];
         if (this_diff < 0) this_diff = -1.0*this_diff;
         if (this_diff>threshold)
          { numdiffs++;
            if (this_diff > maxdiff) maxdiff=this_diff;
          }
        }
        if (numdiffs > 0)
          printf("%d Diffs found over threshold %f; Max Diff = %f\n",
           numdiffs,threshold,maxdiff);
        else
          printf("No differences found between reference and test versions\n");
      }

      double rtclock(void)
      {
        struct timezone Tzp;
        struct timeval Tp;
        int stat;
        stat = gettimeofday (&Tp, &Tzp);
        if (stat != 0) printf("Error return from gettimeofday: %d",stat);
        return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
      }
