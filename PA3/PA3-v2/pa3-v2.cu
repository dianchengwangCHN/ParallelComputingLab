// Template for Programming Assignment 3
// Use "module load cuda" to enable compilation with the Nvidia C compiler nvcc
// Use "nvcc -O3" to compile code; this can be done even on an OSC login node (does not have a GPU)
// To execute compiled code, you must either use a batch submission to run on a node with GPU
// or obtain an interactive GPU-node by using: qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1 -A PAS1488


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-8
#define n (1024)
// Change n to 1024 for final testing; 
// #define n (256)
// n is set to 256 since execution time of single thread template version is excessive
#define SIZE (32)
void init(void);
void ref(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *C);
double rtclock(void);

double a[n][n],b[n][n],c[n][n],cref[n][n];

int main(){

  double clkbegin, clkend, t;
  double *Ad,*Bd,*Cd;
  int size;

  printf("Matrix Size = %d\n",n);

  init();
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; cref[n/2][n/2-1] = %f; \n",
    2.0*n*n*n/t/1e9,t,cref[n/2][n/2-1]);

  
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Cd,size);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);

  dim3 threads(SIZE, SIZE);
  dim3 grid(n / threads.x, n / threads.y);

  clkbegin = rtclock();
  test_kernel<<<grid, threads>>>(n,Ad,Bd,Cd);
  if (hipDeviceSynchronize() != hipSuccess) 
    printf ("Error return for test_kernel: Was execution done on a node with a GPU?\n");
  else
  {
   clkend = rtclock();
   t = clkend-clkbegin;
   hipMemcpy(c,Cd,size,hipMemcpyDeviceToHost);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);
   printf("GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; c[n/2][n/2-1] = %f; \n",
     2.0*n*n*n/t/1e9,t,c[n/2][n/2-1]);
   printf("Correctness Check for GPU solution:\n");
   compare(n, (double *) c,(double *) cref);
 }
}

__global__ void test_kernel(int N, double *A, double *B, double *C)
{
  int i,j,k;
// Template version uses only one thread, which does all the work
// This must be changed (and the launch parameters) to exploit GPU parallelism
// You can make any changes; only requirement is that correctness test passes

  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;

  int tx = threadIdx.x, ty = threadIdx.y;
  __shared__ double Bb[SIZE * SIZE], Ab[SIZE * SIZE];

  double sum = 0;
  for(int ks = 0; ks < N; ks += SIZE){
    Bb[ty * SIZE + tx] = B[(ks + tx) * N + j];
    Ab[ty * SIZE + tx] = A[(ks + ty) * N + i];
    __syncthreads();
    for(k = 0; k < SIZE; k++){
      sum += Ab[k * SIZE + tx] * Bb[ty * SIZE + k];
    }
    __syncthreads();
  }
  C[j * N + i] = sum;
}

void ref(void)
{
  int i,j,k;

  for (j=0;j<n;j++)
   for (k=0;k<n;k++)
    for (i=0;i<n;i++)
      cref[j][i] += a[k][i]*b[k][j];
  }

  void init(void)
  {
    int i,j;
    for(i=0;i<n;i++)
     for(j=0;j<n;j++) 
       { c[i][j] = 0.0; 
         cref[i][j] = 0.0; 
         a[i][j] = drand48();
         b[i][j] = drand48();
       }
     }

     void compare(int N, double *wref, double *w)
     {
      double maxdiff,this_diff;
      int numdiffs;
      int i,j;
      numdiffs = 0;
      maxdiff = 0;
      for (i=0;i<N;i++)
       for (j=0;j<N;j++)
       {
         this_diff = wref[i*N+j]-w[i*N+j];
         if (this_diff < 0) this_diff = -1.0*this_diff;
         if (this_diff>threshold)
          { numdiffs++;
            if (this_diff > maxdiff) maxdiff=this_diff;
          }
        }
        if (numdiffs > 0)
          printf("%d Diffs found over threshold %f; Max Diff = %f\n",
           numdiffs,threshold,maxdiff);
        else
          printf("No differences found between reference and test versions\n");
      }

      double rtclock(void)
      {
        struct timezone Tzp;
        struct timeval Tp;
        int stat;
        stat = gettimeofday (&Tp, &Tzp);
        if (stat != 0) printf("Error return from gettimeofday: %d",stat);
        return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
      }
